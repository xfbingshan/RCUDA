
#include <hip/hip_runtime.h>
#define NUM_THREADS 32
#define size_t int

extern "C"
__global__ void 
euclidean_kernel(const float * vg_a, size_t pitch_a, size_t n_a,
		 const float * vg_b, size_t pitch_b, size_t n_b,
		 size_t k,
		 float * d, size_t pitch_d,
		 float p)
{
  size_t x = blockIdx.x;
  size_t y = blockIdx.y;

  // If an element is to be computed
  if(x < n_a && y < n_b) {

    __shared__ float temp[NUM_THREADS];

    temp[threadIdx.x] = 0.0;
    for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
      float t = vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset];
      temp[threadIdx.x] += (t * t);
    }
    
    // Sync with other threads
    __syncthreads();
    
    // Reduce
    for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
      if(threadIdx.x < stride) {
	temp[threadIdx.x] += temp[threadIdx.x + stride];
      }
      __syncthreads();
    }
    // Write to global memory
    if(threadIdx.x == 0) {
      d[y * pitch_d + x] = sqrt(temp[0]);
    }
  }
}

extern "C"
__global__ void 
euclidean_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a,
		      const float * vg_b, size_t pitch_b, size_t n_b,
		      size_t k, float * d, size_t pitch_d, float p)
{
	size_t x = blockIdx.x, y = blockIdx.y;

	if((x == y) && (x < n_a) && (threadIdx.x == 0))
		d[y * pitch_d + x] = 0.0;
  
	// If all element is to be computed
	if(y < n_a && x < y) {
		__shared__ float temp[NUM_THREADS];    

		temp[threadIdx.x] = 0.0;
    
		for(size_t offset = threadIdx.x; offset < k; offset += NUM_THREADS) {
			float t = vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset];
			temp[threadIdx.x] += (t * t);
		}
    
		// Sync with other threads
		__syncthreads();
    
                // http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html
		// Reduce
		// This is a cumsum.
		// Vital that __syncthreads is called so all threads
                // update temp[threadIdx.x]
		for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
			if(threadIdx.x < stride)
				temp[threadIdx.x] += temp[threadIdx.x + stride];
			__syncthreads();
		}
	    
		// Write to global memory
		if(threadIdx.x == 0) {
			float s = sqrt(temp[0]);
			d[y * pitch_d + x] = s;
			d[x * pitch_d + y] = s;
		}
	}
}
