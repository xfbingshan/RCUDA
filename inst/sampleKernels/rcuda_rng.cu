#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C"
{

// .cuda(setup_kernel, .cu_rng_states, rng_seeds, n_states, gridDim=gridDim, blockDim=blockDim, outputs=NULL)

__global__ void setup_kernel(hiprandState  *state, int *seeds, int n)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    if (idx < n){
        hiprand_init(seeds[idx], idx, 0, &state[idx]);
    }
    return;
}

}


